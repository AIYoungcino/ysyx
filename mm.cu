#include <iostream>
#include <hip/hip_runtime.h>

#define N 512

__global__ void matrixMultiply(float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;
    for (int i = 0; i < N; i++) {
        sum += A[row * N + i] * B[i * N + col];
    }

    C[row * N + col] = sum;
}

int main() {
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    h_A = new float[N * N];
    h_B = new float[N * N];
    h_C = new float[N * N];

    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / 16, N / 16);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Kernel ran for " << milliseconds << " milliseconds" << std::endl;

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Print or further process result matrix h_C

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}